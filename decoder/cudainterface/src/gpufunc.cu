
#include "gpufunc.h"
#include <hip/hip_runtime.h>
#include "bmi_dwt_gpu_kernel.h"


int		GPU_NUMBER	=	182;

	inline void	__bmi_cuda_call(hipError_t err, const char *file, const int line)
	{
		if(err != hipSuccess) {

			printf(
				"ERROR: CUDA Runtime error in file <%s>, line %i:  %s\n",
				file, line, hipGetErrorString(err) );
#if !_DEBUG
			exit(-1);
#else
			assert(0);
#endif
		}
	}

	inline	void	__bmi_cuda_call_ign(hipError_t err, const char *file, const int line)
	{
		if(err != hipSuccess) {
			// reset error state
			err = hipGetLastError();
// 			bmi_eprintf((
// 				"WARNING:  CUDA Runtime error in file <%s>, line %i:  %s\n",
// 				file, line, hipGetErrorString(err) )
// 				);
		}
	}

////////////////////////////////////////////////////////////////////////////////
/////////		Internal functions			///////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
#define BMI_CEILINT(x,y) ((x/y) + ((x%y) ? 1 : 0))

int bmi_idwt_calc_row_grid(dim3 *grid, dim3 *block, int length, int rows)
{
	// set block size to tile width
	block->x = IDWT_TILE_WIDTH;
	block->y = IDWT_TILE_WIDTH;
	// set the grid dimension based on the block dimension
	grid->x = BMI_CEILINT(BMI_CEILDIV2(length), block->x);
	grid->y = BMI_CEILINT(rows, block->y);
	return 0;
}
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

int	bmi_get_gpu_info(GpuInfo *gpu)
{
	// locals
	int count;
	int dev;
	hipDeviceProp_t prop;
	// 	char format_s[] = "  %-20s  %s\n";
	// 	char format_d[] = "  %-20s  %d\n";

	// get the number of devices.  for now, we'll only support one.
	bmi_cuda_call_ign(hipGetDeviceCount(&count));
	if (count == 0) { //***UUU No ! this is not the way hipGetDeviceCount() works
//		PRIN(("Warning:  CUDA not supported by GPUs in this system\n\n"));
		gpu->device = -1;
		return -1;
	}
	if (count > 1) {
//		bmi_eprintf(("Warning:  CUDA supported on %d devices in this system.  Only device 0 will be used.\n\n"));
		count = 1;
	}

	// get the info for devices
	assert(count == 1);
	for (dev = 0; dev < count; dev++)
	{
		if(hipGetDeviceProperties(&prop, dev) == hipSuccess)
		{
			break;
		}
	}
	
	if(prop.major == 9999)//***UUUU  No cuda enabled devices available
	{
		gpu->device = -1;
		return -1;
	}
	

	assert(dev != count);
 	gpu->major = prop.major;
 	gpu->minor = prop.minor;
 	strcpy(gpu->name, prop.name);
	gpu->device = dev;	
	gpu->multiProcessorCount = prop.multiProcessorCount;
	GPU_NUMBER = (gpu->multiProcessorCount<<3) ; // * 8 */
	/*
	Individual GPU program launches are limited to a run time of less than 5 seconds on 
	the device. Exceeding this time limit usually causes a launch failure reported through 
	the CUDA driver or the CUDA runtime, but in some cases hangs the entire machine, 
	requiring a hard reset. Microsoft Windows has a "watchdog" timer that causes programs 
	using the primary graphics adapter to time out if they run longer than the maximum 
	allowed time. For this reason it is recommended that CUDA is run on a G80 that is NOT 
	attached to a display and does not have the Windows desktop extended onto it. In this 
	case, the system must contain at least one NVIDIA GPU that serves as the primary graphics 
	adapter.

	SHIT!!! if we reserve one processor, 10-msec slower!
	*/
	gpu->clockRate = prop.clockRate;
	gpu->memory = (int)(prop.totalGlobalMem);
	// set device
	gpu->active = 1;
	bmi_cuda_call_ign(hipSetDevice(gpu->device));

	// check for any miscellaneous errors hanging around
	bmi_cuda_call_ign(hipDeviceSynchronize());

	return 0;
}

int	bmi_del_gpu_info(GpuInfo *gpu) 
{
	// just reset the thread state
	bmi_cuda_call(hipDeviceReset());
	gpu->active = 0;
	return 0;
}


void malloc_gpu_buf(CudaBuf * buf)
{

	assert(buf->mBuf == NULL);

	switch(buf->mType)
	{
	case GPU_BUF:
		if (buf->mWidth != 0 && buf->mHeight != 0)
		{
			buf->mSize = buf->mWidth * buf->mHeight;
		}
		buf->mPitch = buf->mWidth;
		bmi_cuda_call(hipMalloc(&buf->mBuf, buf->mSize));
		break;

	case GPU_ALLIGN_BUF:
		bmi_cuda_call(hipMallocPitch(&buf->mBuf, (size_t *)&buf->mPitch, (size_t)buf->mWidth, (size_t)buf->mHeight));
		buf->mSize = buf->mPitch * buf->mHeight;
		break;

	case HOST_ALLLIGN_BUF:
	case HOST_BUF:
		buf->mSize = buf->mWidth * buf->mHeight;
		buf->mPitch = buf->mWidth;
		bmi_cuda_call(hipHostMalloc(&buf->mBuf, buf->mSize));
		break;
	default:
		assert(0);
		break;
	}

}

void delete_gpu_buf(CudaBuf * buf)
{

	switch(buf->mType)
	{
	case GPU_BUF:
	case GPU_ALLIGN_BUF:
		bmi_cuda_call(hipFree(buf->mBuf));
		break;

	case HOST_ALLLIGN_BUF:
	case HOST_BUF:
		bmi_cuda_call(hipHostFree(buf->mBuf));
		break;

	default:
		assert(0);
		break;
	}
}

StreamId bmi_gpu_stream_new()
{
	StreamId ret = 0;
	bmi_cuda_call(hipStreamCreate(&ret));
	return ret;
}

void bmi_gpu_stream_delete(StreamId  * P_gpu_str)
{
	// clean up stream context
	if (P_gpu_str)
	{
		bmi_cuda_call(hipStreamDestroy(* P_gpu_str));
	}

}


void bmi_sync_job(StreamId * P_asyncId)
{
	if (P_asyncId)
	{
		bmi_cuda_call(hipStreamSynchronize(*P_asyncId));
	}
	else	// sync all unfinished job
	{
		bmi_cuda_call(hipDeviceSynchronize());
	}
}

int memory_copy_linear(void * dest_buf, void * src_buf, int copy_size_in_byte, StreamId async_id, int direction)
{
	int dir;
	if		(direction == CUDA_COPY_HOST_TO_DEVICE)		dir = hipMemcpyHostToDevice;
	else if (direction == CUDA_COPY_HOST_TO_HOST)		dir = hipMemcpyHostToHost;
	else if (direction == CUDA_COPY_DEVICE_TO_DEVICE)	dir = hipMemcpyDeviceToDevice;
	else if (direction == CUDA_COPY_DEVICE_TO_HOST)		dir = hipMemcpyDeviceToHost;
	else	assert(0);

	if (async_id > 0)
	{
		bmi_cuda_call(hipMemcpyAsync( dest_buf,  src_buf, copy_size_in_byte, (hipMemcpyKind)dir, async_id ) );
	}
	else
	{
		bmi_cuda_call(hipMemcpy( dest_buf,  src_buf, copy_size_in_byte, (hipMemcpyKind)dir ) );
	}
	return 0;

}

int memory_copy_2D(void * dest_buf, size_t dest_pitch, void * src_buf,   size_t src_pitch, size_t width, size_t height, StreamId *  P_asyncId, int direction  )
{

	int dir;
	if		(direction == CUDA_COPY_HOST_TO_DEVICE)		dir = hipMemcpyHostToDevice;
	else if (direction == CUDA_COPY_HOST_TO_HOST)		dir = hipMemcpyHostToHost;
	else if (direction == CUDA_COPY_DEVICE_TO_DEVICE)	dir = hipMemcpyDeviceToDevice;
	else if (direction == CUDA_COPY_DEVICE_TO_HOST)		dir = hipMemcpyDeviceToHost;
	else	assert(0);

	if (P_asyncId)
	{
		bmi_cuda_call(hipMemcpy2DAsync( dest_buf,  dest_pitch, src_buf,  src_pitch,  width,  height, (hipMemcpyKind)dir, * P_asyncId ) );
	}
	else
	{
 		bmi_cuda_call(hipMemcpy2D( dest_buf,  dest_pitch, src_buf,  src_pitch,  width,  height, (hipMemcpyKind)dir));
	}
	return 0;
}

// 
// int memory_copy_device_to_host_2D (void * host_buf, void * device_buf,  size_t host_pitch, size_t device_pitch, size_t width, size_t height, StreamId  async_id)
// {
// 	if (async_id > 0)
// 	{
// 		bmi_cuda_call(hipMemcpy2DAsync( host_buf,  host_pitch, device_buf,  device_pitch,  width,  height, hipMemcpyDeviceToHost, async_id ) );
// 	}
// 	else
// 	{
//  		bmi_cuda_call(hipMemcpy2D(host_buf,  host_pitch, device_buf,  device_pitch,  width,  height, hipMemcpyDeviceToHost));
// 	}
// 	return 0;
// }
// 
// int memory_copy_device_to_device_2D( void * dest_buf, void * src_buf,  size_t dest_pitch, size_t src_pitch, size_t width, size_t height, StreamId  async_id)
// {
// 		if (async_id > 0)
// 	{
// 		bmi_cuda_call(hipMemcpy2DAsync( dest_buf,  dest_pitch, src_buf,  src_pitch,  width,  height, hipMemcpyDeviceToDevice, async_id ) );
// 	}
// 	else
// 	{
//  		bmi_cuda_call(hipMemcpy2D(dest_buf,  dest_pitch, src_buf,  src_pitch,  width,  height, hipMemcpyDeviceToDevice));
// 	}
// 	return 0;
// }
// 
// 
// int memory_copy_host_to_device_2D( void * host_buf, void * device_buf,  size_t host_pitch, size_t device_pitch, size_t width, size_t height, StreamId  async_id)
// {
// 	if (async_id > 0)
// 	{
// 		bmi_cuda_call(hipMemcpy2DAsync( device_buf,  device_pitch, host_buf,  host_pitch,  width,  height, hipMemcpyHostToDevice, async_id ) );
// 	}
// 	else
// 	{
//  		bmi_cuda_call(hipMemcpy2D(device_buf,  device_pitch, host_buf,  host_pitch,  width,  height, hipMemcpyHostToDevice));
// 	}
// 	return 0;
// }

// int	 bmi_i97_dequantize_component(CudaBuf * deviceBuf, TileInfo_c * tileInfo, int compId, short wordShift, StreamId asyncId)
// {
// 		// 	dim3 block_size;
// 
// 	int blocknum = tileInfo->mSize.y;
// 
// 	int threadnum = GPU_NUMBER < tileInfo->mSize.x ? GPU_NUMBER :  tileInfo->mSize.x;
// 
// 	if (asyncId <= 0)
// 	{
// 		bmi_i97_dequantize_transfer<<<blocknum, threadnum, 0 >>>(
// 				(int *)deviceBuf->mBuf,
// 				tileInfo->mOff.x,
// 				tileInfo->mOff.x,
// 				BytesToPixels(deviceBuf->mPitch, wordShift),
// 				tileInfo->mSize.x,
// 				tileInfo->mSize.y,
// 				subbandInfo->mAbsStep,
// 				threadnum,
// 				subbandInfo->mQquanShift
// 				);	
// 	}
// 	else
// 	{
// 		bmi_i97_dequantize_transfer<<<blocknum, threadnum, 0, asyncId >>>(
// 				(int *)deviceBuf->mBuf,
// 				subbandInfo->mOff.x + tile_off_x,
// 				subbandInfo->mOff.y + tile_off_y,
// 				BytesToPixels(deviceBuf->mPitch, wordShift),
// 				subbandInfo->mSize.x,
// 				subbandInfo->mSize.y,
// 				subbandInfo->mAbsStep,
// 				threadnum,
// 				subbandInfo->mQquanShift
// 				);	
// 	}
// 	
// 
// 	return 0;
// }

#if !GPU_W9X7_FLOAT
int	 bmi_i97_dequantize_subband(CudaBuf * deviceBuf, int tile_off_x, int tile_off_y, SubbandInfo_c * subbandInfo,short wordShift,  StreamId asyncId)
{
	// 	dim3 block_size;

	int blocknum = subbandInfo->mSize.y;

	int threadnum = GPU_NUMBER < subbandInfo->mSize.x ? GPU_NUMBER :  subbandInfo->mSize.x;

	if (asyncId <= 0) 
	{
		bmi_i97_dequantize_transfer<<<blocknum, threadnum, 0 >>>( //***UUU use <<<blocknum, threadnum, 0 , (asyncId <= 0 ? 0 : asyncId) >>>
				(int *)deviceBuf->mBuf,
				subbandInfo->mOff.x + tile_off_x,
				subbandInfo->mOff.y + tile_off_y,
				BytesToPixels(deviceBuf->mPitch, wordShift),
				subbandInfo->mSize.x,
				subbandInfo->mSize.y,
				subbandInfo->miAbsStep,
				threadnum,
				subbandInfo->mQquanShift
				);	
	}
	else //****UUU This repitition is not needed, one for asyncId = 0 and asyncId > 0
	{
		bmi_i97_dequantize_transfer<<<blocknum, threadnum, 0, asyncId >>>(
				(int *)deviceBuf->mBuf,
				subbandInfo->mOff.x + tile_off_x,
				subbandInfo->mOff.y + tile_off_y,
				BytesToPixels(deviceBuf->mPitch, wordShift),
				subbandInfo->mSize.x,
				subbandInfo->mSize.y,
				subbandInfo->miAbsStep,
				threadnum,
				subbandInfo->mQquanShift
				);	
	}
	

	return 0;
}
#endif

int	 bmi_dc_shift(CudaBuf * compBuf, CudaBuf * compBufTemp, int src_x_off, int src_y_off, int dst_x_off, int dst_y_off, int widthInPixel, int heightInpixel, int pixelWordLength, int bit_depth, int isFloat, StreamId asyncId)
{
	// 	dim3 block_size;

	int blocknum = heightInpixel;

	int threadNum = widthInPixel;
	while(threadNum > MAX_THREADS)
	{
		threadNum = (threadNum + 1) /2;
	}

		bmi_dc_shift_kernel<<<blocknum, threadNum, 0,(asyncId <= 0 ? 0 : asyncId) >>>( //***UUU use <<<blocknum, threadnum, 0 , (asyncId <= 0 ? 0 : asyncId) >>>
				(int *)compBuf->mBuf,
				(int *)compBufTemp->mBuf,
				src_x_off,
				src_y_off,
				dst_x_off,
				dst_y_off,
				(compBuf->mPitch / pixelWordLength),
				(compBuf->mPitch / pixelWordLength),//use the same pitch
				widthInPixel,
				threadNum,
				isFloat, 
				bit_depth
				);	
	
	

	return 0;
}


void bmi_idwt_r53_gpu_transform(TileInfo_c * tile,CompInfo_c	* comp, CudaBuf * orig, CudaBuf * temp, StreamId asyncId, short wordShift, int toplevel)
{

// 	printf("doing DWT for a component , use async %d\n", asyncId);
	// kernel launch variables
	dim3 block_size;
	dim3 grid_size;
	int tile_off_y = tile->mOff.y;
	int tile_off_x = tile->mOff.x;
	int dwtLevel = (comp->mNumOfSubband - 1) /3 ;
	if (toplevel != -1)
	{
		dwtLevel = (dwtLevel > toplevel ? toplevel : dwtLevel);
	}


	unsigned int  texture_orig_size = orig->mPitch * tile_off_y;
	// now it's the y offset

	unsigned char * texture_orig_buf = (unsigned char *)orig->mBuf + texture_orig_size /*+ PixelsToBytes(tile_off_x, wordShift)*/;
	texture_orig_size = 	orig->mSize - texture_orig_size;
	// now we got the real size
		
	unsigned int  texture_temp_size = temp->mPitch * tile_off_x;
	// now it's the offset

	unsigned char * texture_temp_buf = (unsigned char *)temp->mBuf + texture_temp_size /*+ PixelsToBytes(tile_off_y, wordShift)*/;
	texture_temp_size = 	temp->mSize - texture_temp_size;
	// now we got the real size

	// iterate over all the levels.  each call of the kernel processes a row
	// and transposes the data.  each level requires two passes.


	const struct textureReference* texRefPtr; //***UUU
	hipGetTextureReference(&texRefPtr, HIP_SYMBOL("tc_tex_int")); //***UUUU
	size_t a=0; //****UUUU
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindSigned); //***UUU
	
	for (int level = 0; level < dwtLevel; level++)
	{
		// shorthand variables
		// - pitch info
		int d_pitch  = BytesToPixels(orig->mPitch, wordShift);
		int t_pitch  = BytesToPixels(temp->mPitch, wordShift);
		// - row info
		int r_par    = tile->mXParity[level] ; /*comp->mSubbandInfo[level * 3 + 2].mOff.x;*/
		int r_len    = comp->mSubbandInfo[level * 3 + 2].mSize.x + comp->mSubbandInfo[level * 3 + 3].mSize.x;
		int r_len_lo = comp->mSubbandInfo[level * 3 + 2].mSize.x;
		// - column info
		int c_par    = tile->mYParity[level] ; /*comp->mSubbandInfo[level * 3 + 1].mOff.y; */
		int c_len    = comp->mSubbandInfo[level * 3 + 1].mSize.y + comp->mSubbandInfo[level * 3 + 3].mSize.y;
		int c_len_lo = comp->mSubbandInfo[level * 3 + 1].mSize.y;

		// launch the kernel on the rows
		

		bmi_idwt_calc_row_grid(&grid_size, &block_size, r_len, c_len);
		bmi_cuda_call(hipBindTexture(&a, texRefPtr/*tc_tex_int*/, (void *)texture_orig_buf, &channelDesc, texture_orig_size));
		//assert(a == 0); //***UUU remove &a, texRefPtr and &channelDesc
		if (wordShift == 2 )	// INT_32_BITS
		{
			if (asyncId <= 0)
			{
				bmi_idwt_r53_1d_gpu_row_int32<<<grid_size, block_size, 0>>>
					((int *)texture_temp_buf + tile_off_y, tile_off_x, t_pitch, d_pitch, r_len, r_len_lo, r_par, c_len);

			}
			else
			{
				bmi_idwt_r53_1d_gpu_row_int32<<<grid_size, block_size, 0>>>
					((int *)texture_temp_buf + tile_off_y, tile_off_x, t_pitch, d_pitch, r_len, r_len_lo, r_par, c_len);

			}
		}
		else
		{
			assert(0); // not support yet
// 			assert(wordShift == 1 );	// SHORT_16_BITS
// 
// 			if (asyncId <= 0)
// 			{
// 				bmi_idwt_r53_1d_gpu_row_short16<<<grid_size, block_size, 0>>>
// 					((short *)temp->mBuf, 0,t_pitch, d_pitch, r_len, r_len_lo, r_par, c_len, true);
// 			}
// 			else
// 			{
// 				bmi_idwt_r53_1d_gpu_row_short16<<<grid_size, block_size, 0, asyncId>>>
// 					((short *)temp->mBuf, 0,t_pitch, d_pitch, r_len, r_len_lo, r_par, c_len, true);
// 			}

		}

		// launch the kernel on the columns
		bmi_idwt_calc_row_grid(&grid_size, &block_size, c_len, r_len);
		bmi_cuda_call(hipBindTexture(0, tc_tex_int, (void *)texture_temp_buf, texture_temp_size));
		if (wordShift == 2 )	// INT_32_BITS
		{
			if (asyncId <= 0)
			{
				bmi_idwt_r53_1d_gpu_row_int32<<<grid_size, block_size, 0>>>
					((int *)texture_orig_buf + tile_off_x,tile_off_y,  d_pitch, t_pitch, c_len, c_len_lo, c_par, r_len);

			}
			else
			{
				bmi_idwt_r53_1d_gpu_row_int32<<<grid_size, block_size, 0, asyncId>>>
					((int *)texture_orig_buf + tile_off_x,tile_off_y,d_pitch, t_pitch, c_len, c_len_lo, c_par, r_len);
			}

		}
		else
		{
			assert(0); // not support yet
// 			assert(wordShift == 1 );	// SHORT_16_BITS
// 			if (asyncId <= 0)
// 			{
// 				bmi_idwt_r53_1d_gpu_row_short16<<<grid_size, block_size, 0>>>
// 				((short *)orig->mBuf, 0,d_pitch, t_pitch, c_len, c_len_lo, c_par, r_len, false);
// 			}
// 			else
// 			{
// 				bmi_idwt_r53_1d_gpu_row_short16<<<grid_size, block_size, 0, asyncId>>>
// 				((short *)orig->mBuf,0,d_pitch, t_pitch, c_len, c_len_lo, c_par, r_len, false);
// 			}
		}
	}
}


void 	SetGridSize(dim3 *grid_size, dim3 * block_size, int lines, int rows)
{

	int SM = GPU_NUMBER / 8;
	grid_size->z = 1;
	if (lines <= SM)
	{
		grid_size->x = lines;
		grid_size->y = 1;
	}
	else
	{
		grid_size->x = SM;
		grid_size->y =(lines + SM - 1) / SM;
	}
	// lineId = grid_size.y * SM + grid_size.x

	int lineLength = rows;
	while (((lineLength + 2) * 4 ) > 16000)	// count't store a line in one SM shared memory
		// 2 : we need 2 random data at the front /end of the line
	{
		++grid_size->z;
		lineLength = (rows + grid_size->z - 1) / grid_size->z;
	}
	// grid_z used for calculate line_offset

	block_size->z = 1;
	if (lineLength <= 64)	// 64 = 32 * 2 :32  threads number of each warp
	{
		block_size->x = (lineLength + 1) / 2;
		block_size->y = 1;
	}
	else
	{
		block_size->x = 32;		// threads number of each warp
		lineLength = (lineLength + 1 ) / 2;	// each thread handle 2 pixels
		// w_num * 32 * loop >= lineLength	: w_num: warp number, less than 24,  loop : each thread loop number 
		// w_num * loop >= lineLength / 32
		int loop = 1;

		lineLength = (lineLength + 31) / 32;
// 		int w_num;
		do 
		{
			block_size->y = (lineLength + loop - 1) / loop;
			++loop;
		} while (block_size->y > 24);
// 		block_size->y = w_num;
	}

	// threadId = y * 32 + x
}

void bmi_idwt_r53_gpu_transform_resolution(CudaBuf * orig, CudaBuf * temp,				// input suda buffer and temp cuda buffer used for DWT
										   int x_off, int y_off,						// x-offset and y-offset in input buffer
										   int lowW, int highW, int lowH, int highH,		// the width and height for the 4 subbands in this resotion
										   int xParity, int yParity,					// the parity for vertial and horizontal
										   int	isLLBandOrig,								// Is this dwt the first DWT? 
										   short wordlength, StreamId asyncId)			// wordlength of input buffer and the asyncId
{
#pragma BMI_PRAGMA_TODO_MSG(("Do DWT for W5X3 by resolution"))
	assert(0);

}


void bmi_idwt_i97_gpu_transform_resolution(CudaBuf * orig, CudaBuf * temp,				// input suda buffer and temp cuda buffer used for DWT
										   int x_off, int y_off,						// x-offset and y-offset in input buffer
										   int lowW, int highW, int lowH, int highH,		// the width and height for the 4 subbands in this resotion
										   int xParity, int yParity,					// the parity for vertial and horizontal
										   float coef0, float coef1, float coef2, float coef3,	// the 4 coefiient for dwt calculation, for original data will be NORM * qqStep * logain/hiGain;
																								// for data from last level will be logain / highgain
										   int	isLLBandFloat,								// Is this dwt the first DWT? if yes will do the RENORM transfer to int
										   short wordlength, StreamId asyncId)			// wordlength of input buffer and the asyncId
{
#if 1
	{

		// buffer pointers
		// kernel launch variables
		dim3 block_size;
		dim3 grid_size;


		unsigned int  texture_orig_size = orig->mPitch * y_off;
		// now it's the y offset

		unsigned char * texture_orig_buf = (unsigned char *)orig->mBuf + texture_orig_size /*+ PixelsToBytes(tile_off_x, wordShift)*/;
		texture_orig_size = 	orig->mSize - texture_orig_size;
		// now we got the real size

		unsigned int  texture_orig_sizeH = orig->mPitch * (y_off + lowH);
		// now it's the y offset

		unsigned char * texture_orig_bufH = (unsigned char *)orig->mBuf + texture_orig_sizeH /*+ PixelsToBytes(tile_off_x, wordShift)*/;
		texture_orig_sizeH = 	orig->mSize - texture_orig_sizeH;

		unsigned int  texture_temp_size = temp->mPitch * x_off;
		// now it's the offset
		unsigned char * texture_temp_buf = (unsigned char *)temp->mBuf + texture_temp_size /*+ PixelsToBytes(tile_off_y, wordShift)*/;
		texture_temp_size = 	temp->mSize - texture_temp_size;
		// now we got the real size



			// - pitch info
			int d_pitch  = orig->mPitch / 4;
			int t_pitch  = temp->mPitch / 4;
			// - row info

			int r_len    = lowW + highW;
			int r_len_lo = lowW;
			// - column info
			int c_len    = highH + lowH ;
			int c_len_lo = lowH;

			// launch the kernel on the rows

			if (asyncId <= 0)
			{
			bmi_idwt_calc_row_grid(&grid_size, &block_size, r_len, lowH);
			bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_buf, texture_orig_size));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_temp_buf + y_off, x_off, t_pitch, d_pitch, r_len, r_len_lo, xParity, lowH, coef0, coef1, isLLBandFloat,0);
			bmi_idwt_calc_row_grid(&grid_size, &block_size, r_len, highH);
			bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_bufH, texture_orig_sizeH));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_temp_buf + y_off + lowH, x_off, t_pitch, d_pitch, r_len, r_len_lo, xParity, highH, coef2, coef3,0,0);
			}
			else
			{
			bmi_idwt_calc_row_grid(&grid_size, &block_size, r_len, lowH);
			bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_buf, texture_orig_size));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0, asyncId>>>
					((float *)texture_temp_buf + y_off, x_off, t_pitch, d_pitch, r_len, r_len_lo, xParity, lowH, coef0, coef1, isLLBandFloat, 0);
			bmi_idwt_calc_row_grid(&grid_size, &block_size, r_len, highH);
			bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_bufH, texture_orig_sizeH));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_temp_buf + y_off + lowH, x_off, t_pitch, d_pitch, r_len, r_len_lo, xParity, highH, coef2, coef3,0, 0);
			}


			// launch the kernel on the columns
			bmi_idwt_calc_row_grid(&grid_size, &block_size, c_len, r_len);
			bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_temp_buf, texture_temp_size));
			//			bmi_cuda_call(hipBindTexture(0, tc_tex_int, temp->mBuf, temp->mSize));

			if (asyncId <= 0)
			{
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_orig_buf + x_off,y_off,  d_pitch, t_pitch, c_len, c_len_lo, yParity, r_len, IDWT_LO_GAIN, IDWT_HI_GAIN, 1, 1);
			}
			else
			{
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0, asyncId>>>
					((float *)texture_orig_buf + x_off, y_off,  d_pitch, t_pitch, c_len, c_len_lo, yParity, r_len, IDWT_LO_GAIN, IDWT_HI_GAIN,1, 1);
			}

		return;
	}
#else
	{


#define BYTES2PIXELS(x)		((x)/4)
#define PIXELS2BYTES(x)		((x)*4)

#define BLOCK_DIM 16

		int  lines = lowH + highH;
		int	 rows = lowW + highW;

		// assume each thread handle one pixels at a time
		// If each block we set 32N threads, calculate for 64N pixels (N is the number of warp, up to 24, 768 threads)
		// like below :  R is random, lower case is loaded form orig and upper case means calcualted data; ' means will be ignore
		// Each block we can calculate at least thredNum * 2 - 8 = 64N - 8 pixels :
		// shared memory we need 2 more pixels for random, size will be (64N + 2) * 4
		// considering for 2K image, 
		// In horizontal DWT, 1080 lines for 2048 pixels, we can set 1080 blocks, each block has 18 * 32 = 576 threadsneed, need 8K+ shard memory, each SM one block
		// each thread calculate 4 pixels, good for 576 * 4 - 8 = 2296 pixels
		// In horizontal DWT, 2048 rows for 1080 pixels, we can set 2048 blocks, each block has 10 * 32 = 320 threadsneed, need 4K+ shard memory, each SM 2 block
		// each thread calculate 4 pixels, good for 320 * 4 - 8 = 1272 pixels

		// 97 no parity:	NORMAL case : throw away 8 pixels, 
		// 	R	h'	l'	h'|	l	h	l	h |	l'	h'	l'	h'	l'	R
		// 			L'	  |	L		L	  |	L'		L'		L'
		// 		H'		H'|		H		H |		H'		H'
		// 			L'	  |	L		L	  |	L'		L'		L'
		// 		H'		H'|		H		H |		H'		H'
		// 97 no parity:odd	AT THE END OF LINE : throw away 7 pixels, 
		// 	R	h'	l'	h'|	l	h	l |	h' 	l'	h''	l'	R
		// 			L'	  |	L		L |	  	L		L''		
		// 		H'		H'|		H	  |	H' 		H'
		// 			L'	  |	L		L |	  	L'		L'
		// 		H'		H'|		H	  |	H' 		H'
		// 97 parity:	AT THE BEGINING OF LINE : throw away 9 pixels, 
		// 	R	h'	l'	h'	l'|	h	l	h |	l'	h'	l'	h'	l'	R
		// 			L'	  	L'|		L	  |	L'		L'		L'
		// 		H'		H'	  |	H		H |		H'		H'
		// 			L'	  	L'|		L	  |	L'		L'		L'
		// 		H'		H'	  |	H		H |		H'		H'

		dim3 block_size;
		dim3 grid_size;


		// 	unsigned int  texture_orig_size = orig->mPitch * y_off;
		// 	// now it's the y offset
		// 
		// 	unsigned char * texture_orig_buf = (unsigned char *)orig->mBuf + texture_orig_size;
		// 	texture_orig_size = 	orig->mSize - texture_orig_size;
		// 	// now we got the real size of the original buffer
		// 		
		// 	unsigned int  texture_temp_size = temp->mPitch * x_off;
		// 	// now it's the offset
		// 	unsigned char * texture_temp_buf = (unsigned char *)temp->mBuf + texture_temp_size;

		float * bufL, * bufH, *dwtResult;
		int calLength;

		// 	texture_temp_size = 	temp->mSize - texture_temp_size;
		// 	// now we got the real size of the temp buffer

		float * tempDBG = (float *)malloc(2048*2048*4 + 256);

		float  * tempBuf  = tempDBG;
		tempBuf = (float *)((int)tempBuf & 0xfffffff0);



		// 		bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_buf, texture_orig_size));
		calLength = rows + (xParity ? 9 : (lowW == highW ? 8 : 7));
		SetGridSize(&grid_size, & block_size, lowH, calLength);
		bufL = (float *)((unsigned char *)orig->mBuf + y_off * orig->mPitch + PIXELS2BYTES(x_off));
		bufH = (float *)((unsigned char *)orig->mBuf + y_off * orig->mPitch + PIXELS2BYTES(x_off + lowW));
		dwtResult = (float *)((unsigned char *)temp->mBuf + x_off * temp->mPitch + PIXELS2BYTES(y_off));
		memory_copy_2D(tempBuf, (lowW+highW)*4,bufL, orig->mPitch, (lowW+highW)*4,lowH+highH, NULL,CUDA_COPY_DEVICE_TO_HOST  );
		gou_dwt97_1d_transfer<<<grid_size, block_size, (lowW+highW+11) * sizeof(float)/*, asyncId*/>>>(
			bufL,				//	float * lowBand,
			bufH,				// float * highBand
			dwtResult,			// float * dwtResult
			orig->mPitch,		// int inPitch
			temp->mPitch,		// int outPitch
			coef0 * IDWT_LO_GAIN, coef1 * IDWT_LO_GAIN,		// float lowCoef, float highCoef
			isLLBandFloat, 0,	//	int isLowBandFloat, int isHighBandFloat,
			lowH,				// int height
			lowW, highW,		// int low_w, int high_w 
			xParity,			// int parity
			grid_size.x,		// int yStep
			block_size.x * block_size.y,		// int x_step
			grid_size.z			// int lineParts
			);		

		memory_copy_2D(tempBuf, lowH*4,dwtResult, temp->mPitch, lowH*4, lowW+highW, NULL,CUDA_COPY_DEVICE_TO_HOST  );



		SetGridSize(&grid_size, & block_size, highH, calLength);
		dwtResult = (float *)((unsigned char *)temp->mBuf + x_off * temp->mPitch + PIXELS2BYTES(y_off + lowH));
		bufL = (float *)((unsigned char *)orig->mBuf + (y_off + lowH) * orig->mPitch + PIXELS2BYTES(x_off));
		bufH = (float *)((unsigned char *)orig->mBuf + (y_off + lowH) * orig->mPitch + PIXELS2BYTES(x_off + lowW));
		gou_dwt97_1d_transfer<<<grid_size, block_size, (lowW+highW+11) * sizeof(float)/*, asyncId*/>>>(
			bufL,				//	float * lowBand,
			bufH,				// float * highBand
			dwtResult,			// float * dwtResult
			orig->mPitch,		// int inPitch
			temp->mPitch,		// int outPitch
			coef2 * IDWT_HI_GAIN, coef3 * IDWT_HI_GAIN,		// float lowCoef, float highCoef
			0, 0,				//	int isLowBandFloat, int isHighBandFloat,
			highH,				// int height
			lowW, highW,		// int low_w, int high_w 
			xParity,			// int parity
			grid_size.x,		// int yStep
			block_size.x * block_size.y,		// int x_step
			grid_size.z			// int lineParts
			);		

		memory_copy_2D(tempBuf+lowH, highH*4,dwtResult, temp->mPitch, highH*4, lowW+highW, NULL,CUDA_COPY_DEVICE_TO_HOST  );

		calLength = lowH + highH + (yParity ? 9 : (lowH == highH ? 8 : 7));
		SetGridSize(&grid_size, & block_size, lowW + highW, calLength);
		dwtResult = (float *)((unsigned char *)orig->mBuf + y_off * orig->mPitch + PIXELS2BYTES(x_off));
		bufL  = (float *)((unsigned char *)temp->mBuf + x_off * temp->mPitch + PIXELS2BYTES(y_off));
		bufH  = (float *)((unsigned char *)temp->mBuf + x_off * temp->mPitch + PIXELS2BYTES(y_off + lowH));
		gou_dwt97_1d_transfer<<<grid_size, block_size, (lowH+highH+11) * sizeof(float)/*, asyncId*/>>>(
			bufL,				//	float * lowBand,
			bufH,				// float * highBand
			dwtResult,			// float * dwtResult
			temp->mPitch,		// int inPitch
			orig->mPitch,		// int outPitch
//			IDWT_LO_GAIN, IDWT_HI_GAIN,			// float lowCoef, float highCoef
			1.0, 1.0,
			1, 1,				//	int isLowBandFloat, int isHighBandFloat,
			lowW + highW,				// int height
			lowH, highH,		// int low_w, int high_w 
			yParity,			// int parity
			grid_size.x,		// int yStep
			block_size.x * block_size.y,		// int x_step
			grid_size.z			// int lineParts
			);		
		memory_copy_2D(tempBuf, 4*(lowW+highW),bufL, orig->mPitch, 4*(lowW+highW),lowH+highH, NULL,CUDA_COPY_DEVICE_TO_HOST  );

		free(tempDBG);
		return;
	}
#endif
}

#if !GPU_W9X7_FLOAT

void bmi_idwt_i97_gpu_transform(TileInfo_c * tile, CompInfo_c	* comp, CudaBuf * orig, CudaBuf * temp, StreamId asyncId, short wordShift, int toplevel)
{
	// buffer pointers
	// kernel launch variables
	dim3 block_size;
	dim3 grid_size;
	int tile_off_y = tile->mOff.y;
	int tile_off_x = tile->mOff.x;


	int dwtLevel = (comp->mNumOfSubband - 1) / 3;

	if (toplevel != -1)
	{
		dwtLevel = (dwtLevel > toplevel ? toplevel : dwtLevel);
	}
	// iterate over all the levels.  each call of the kernel processes a row
	// and transposes the data.  each level requires two passes.

	unsigned int  texture_orig_size = orig->mPitch * tile_off_y;
	// now it's the y offset

	unsigned char * texture_orig_buf = (unsigned char *)orig->mBuf + texture_orig_size /*+ PixelsToBytes(tile_off_x, wordShift)*/;
	texture_orig_size = 	orig->mSize - texture_orig_size;
	// now we got the real size
		
	unsigned int  texture_temp_size = temp->mPitch * tile_off_x;
	// now it's the offset

	unsigned char * texture_temp_buf = (unsigned char *)temp->mBuf + texture_temp_size /*+ PixelsToBytes(tile_off_y, wordShift)*/;
	texture_temp_size = 	temp->mSize - texture_temp_size;
	// now we got the real size


	for (int level = 0; level < dwtLevel; level++) {

		// - pitch info
		int d_pitch  = BytesToPixels(orig->mPitch, wordShift);
		int t_pitch  = BytesToPixels(temp->mPitch, wordShift);
		// - row info
		int r_par    = (int)tile->mXParity[level]; /*comp->mSubbandInfo[level * 3 + 2].mOff.x; */

		int r_len    = comp->mSubbandInfo[level * 3 + 2].mSize.x + comp->mSubbandInfo[level * 3 + 3].mSize.x;
		int r_len_lo = comp->mSubbandInfo[level * 3 + 2].mSize.x;
		// - column info
		int c_par    = (int)tile->mYParity[level]; /*comp->mSubbandInfo[level * 3 + 1].mOff.y; */
		int c_len    = comp->mSubbandInfo[level * 3 + 1].mSize.y + comp->mSubbandInfo[level * 3 + 3].mSize.y;
		int c_len_lo = comp->mSubbandInfo[level * 3 + 1].mSize.y;

		if (wordShift == 2 )	// INT_32_BITS
		{
			// launch the kernel on the rows
			bmi_idwt_calc_row_grid(&grid_size, &block_size, r_len, c_len);
			bmi_cuda_call(hipBindTexture(0, tc_tex_int, (void *)texture_orig_buf, texture_orig_size));

			if (asyncId <= 0)
			{
				bmi_idwt_i97_1d_gpu_row<<<grid_size, block_size, 0>>>
					((int *)texture_temp_buf + tile_off_y, tile_off_x, t_pitch, d_pitch, r_len, r_len_lo, r_par, c_len);
			}
			else
			{
				bmi_idwt_i97_1d_gpu_row<<<grid_size, block_size, 0, asyncId>>>
					((int *)texture_temp_buf + tile_off_y, tile_off_x, t_pitch, d_pitch, r_len, r_len_lo, r_par, c_len);
			}


			// launch the kernel on the columns
			bmi_idwt_calc_row_grid(&grid_size, &block_size, c_len, r_len);
			bmi_cuda_call(hipBindTexture(0, tc_tex_int, (void *)texture_temp_buf, texture_temp_size));
//			bmi_cuda_call(hipBindTexture(0, tc_tex_int, temp->mBuf, temp->mSize));

			if (asyncId <= 0)
			{
				bmi_idwt_i97_1d_gpu_row<<<grid_size, block_size, 0>>>
					((int *)texture_orig_buf + tile_off_x,tile_off_y,  d_pitch, t_pitch, c_len, c_len_lo, c_par, r_len);
			}
			else
			{
				bmi_idwt_i97_1d_gpu_row<<<grid_size, block_size, 0, asyncId>>>
					((int *)texture_orig_buf + tile_off_x,tile_off_y,d_pitch, t_pitch, c_len, c_len_lo, c_par, r_len);
			}

		}
		else
		{
			assert(0); // todo : add 97 16 bits support
		}
	}

}
#else


void bmi_idwt_i97_gpu_transform_float(TileInfo_c * tile, CompInfo_c	* comp, CudaBuf * orig, CudaBuf * temp, StreamId asyncId, short wordShift, int toplevel)
{
	// buffer pointers
	// kernel launch variables
	dim3 block_size;
	dim3 grid_size;


	int dwtLevel = (comp->mNumOfSubband - 1) / 3;

	if (toplevel != -1)
	{
		dwtLevel = (dwtLevel > toplevel ? toplevel : dwtLevel);
	}
	// iterate over all the levels.  each call of the kernel processes a row
	// and transposes the data.  each level requires two passes.


	unsigned char * texture_orig_buf = (unsigned char *)orig->mBuf + orig->mPitch * tile->mOff.y ;
	unsigned int  texture_orig_size = orig->mSize - orig->mPitch * tile->mOff.y;
		
	unsigned char * texture_temp_buf = (unsigned char *)temp->mBuf + temp->mPitch * tile->mOff.x ;
	unsigned int  texture_temp_size = temp->mSize - temp->mPitch * tile->mOff.x;


	for (int level = 0; level < dwtLevel; level++) {

		// - pitch info
		int in_pitch  = BytesToPixels(orig->mPitch, wordShift);
		int out_pitch  = BytesToPixels(temp->mPitch, wordShift);

		int x_par    = tile->mXParity[level] ;
		int y_par    = tile->mYParity[level];

		
		int width    = comp->mSubbandInfo[level * 3 + 2].mSize.x + comp->mSubbandInfo[level * 3 + 3].mSize.x;
		int lowW = comp->mSubbandInfo[level * 3 + 2].mSize.x;

		int height   = comp->mSubbandInfo[level * 3 + 1].mSize.y + comp->mSubbandInfo[level * 3 + 3].mSize.y;
		int lowH     = comp->mSubbandInfo[level * 3 + 1].mSize.y;
		int highH     = comp->mSubbandInfo[level * 3 + 3].mSize.y;

		if (wordShift == 2 )	// INT_32_BITS
		{
			// launch the kernel on the rows

			if (asyncId <= 0)
			{
				// LL and LH
				bmi_idwt_calc_row_grid(&grid_size, &block_size, width, lowH);
				bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_buf, texture_orig_size));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_temp_buf + tile->mOff.y, tile->mOff.x,
					out_pitch, in_pitch, 
					width, lowW, x_par,
					lowH, 
					level ? IDWT_LO_GAIN : comp->mSubbandInfo[0].mfAbsStepNorm * IDWT_LO_GAIN,
					comp->mSubbandInfo[level * 3 + 1].mfAbsStepNorm * IDWT_HI_GAIN,
					level ? 1 : 0,
					0);

				// HL and LL
				bmi_idwt_calc_row_grid(&grid_size, &block_size, width, highH);
				bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)(texture_orig_buf + lowH * orig->mPitch), (texture_orig_size - lowH * orig->mPitch)));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_temp_buf + tile->mOff.y + lowH, tile->mOff.x,
					out_pitch, in_pitch, 
					width, lowW, x_par,
					highH, 
					comp->mSubbandInfo[level * 3 + 2].mfAbsStepNorm * IDWT_LO_GAIN,
					comp->mSubbandInfo[level * 3 + 3].mfAbsStepNorm * IDWT_HI_GAIN,
					0,
					0);
			}
			else
			{
				bmi_idwt_calc_row_grid(&grid_size, &block_size, width, lowH);
				bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_orig_buf, texture_orig_size));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0, asyncId>>>
					((float *)texture_temp_buf + tile->mOff.y, tile->mOff.x,
					out_pitch, in_pitch, 
					width, lowW, x_par,
					lowH, 
					level ? IDWT_LO_GAIN : comp->mSubbandInfo[0].mfAbsStepNorm * IDWT_LO_GAIN,
					comp->mSubbandInfo[level * 3 + 1].mfAbsStepNorm * IDWT_HI_GAIN,
					level ? 1 : 0,
					0);
				bmi_idwt_calc_row_grid(&grid_size, &block_size, width, highH);
				bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)(texture_orig_buf + lowH * orig->mPitch), (texture_orig_size - lowH * orig->mPitch)));
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0, asyncId>>>
					((float *)texture_temp_buf + tile->mOff.y + lowH, tile->mOff.x,
					out_pitch, in_pitch, 
					width, lowW, x_par,
					highH, 
					comp->mSubbandInfo[level * 3 + 2].mfAbsStepNorm * IDWT_LO_GAIN,
					comp->mSubbandInfo[level * 3 + 3].mfAbsStepNorm * IDWT_HI_GAIN,
					0,
					0);			
			}


			// launch the kernel on the columns
			bmi_idwt_calc_row_grid(&grid_size, &block_size, height, width);
			bmi_cuda_call(hipBindTexture(0, tc_tex_float, (void *)texture_temp_buf, texture_temp_size));
			if (asyncId <= 0)
			{
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0>>>
					((float *)texture_orig_buf + tile->mOff.x, tile->mOff.y,
					in_pitch, out_pitch, 
					height, lowH, y_par,
					width, 
					IDWT_LO_GAIN,
					IDWT_HI_GAIN,
					1,
					1);
			}
			else
			{
				bmi_idwt_i97_1d_gpu_row_float<<<grid_size, block_size, 0, asyncId>>>
					((float *)texture_orig_buf + tile->mOff.x, tile->mOff.y,
					 in_pitch, out_pitch,
					height, lowH, y_par,
					width, 
					IDWT_LO_GAIN,
					IDWT_HI_GAIN,
					1,
					1);
			}

		}
		else
		{
			assert(0); // todo : add 97 16 bits support
		}
	}
	
}

#endif

int bmi_MCT_gpu(TileInfo_c * tile, int bitDepth, int wordShift,int dwtLevel,  int decodeLevel,int elementPerPixel, int bitsPerElement, StreamId asyncId)
{

	int threadNum;

	int blocknum = tile->mSize.y; //mCompInfo[0].mCompBuf->mHeight;
	int width_pixel = tile->mSize.x;
	int result_x_off = tile->mOff.x;
	int result_y_off = tile->mOff.y;
	if (decodeLevel < 0 || decodeLevel > dwtLevel)
	{
		decodeLevel = -1;
	}
 	if (decodeLevel != -1)
	{
		blocknum = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.y + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.y;		width_pixel = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.x + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.x;
		while (dwtLevel > decodeLevel)
		{
			result_x_off = (result_x_off + 1) / 2;
			result_y_off = (result_y_off + 1) / 2;
			dwtLevel--;
		}

	}

	threadNum = width_pixel;
	while(threadNum > MAX_THREADS)
	{
		threadNum = (threadNum + 1) /2;
	}

	if (asyncId < 0)		// sync job
	{
		asyncId = 0;
	}


	if (wordShift == 2)	
	{

		bmi_mct_tile_int32<<<blocknum, threadNum, 0, asyncId>>>(
			(int *)tile->mCompInfo[0].mCompBuf->mBuf,
			(int *)tile->mCompInfo[1].mCompBuf->mBuf,
			(int *)tile->mCompInfo[2].mCompBuf->mBuf,
			tile->mOff.x,
			tile->mOff.y,
			result_x_off,
			result_y_off,
			(unsigned char *)tile->mResultBuf->mBuf,		
			BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
			tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
			bitDepth, 
			bitsPerElement, 
			bitsPerElement,
			elementPerPixel,
			width_pixel,
			threadNum);

	}
	else
	{
		assert(0); // no done yet
	}


	return 0;
}

int bmi_ICT_gpu(TileInfo_c * tile, int bitDepth, int wordShift,int dwtLevel, int decodeLevel, int elementPerPixel, int bitsPerElement,  StreamId asyncId)
{
	int threadNum;

	int blocknum = tile->mSize.y; //mCompInfo[0].mCompBuf->mHeight;
	int width_pixel = tile->mSize.x;
	int result_x_off = tile->mOff.x;
	int result_y_off = tile->mOff.y;
	if (decodeLevel < 0 || decodeLevel > dwtLevel)
	{
		decodeLevel = -1;
	}
 	if (decodeLevel != -1)
	{
		blocknum = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.y + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.y;		
		width_pixel = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.x + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.x;
		while (dwtLevel > decodeLevel)
		{
			result_x_off = (result_x_off + 1) / 2;
			result_y_off = (result_y_off + 1) / 2;
			dwtLevel--;
		}

	}



	threadNum = width_pixel;
	while(threadNum > MAX_THREADS)
	{
		threadNum = (threadNum + 1) /2;
	}

	if (asyncId > 0)		// async job
	{
		if (wordShift == 2)	
		{

#if GPU_W9X7_FLOAT
			bmi_ict_tile_float<<<blocknum, threadNum, 0, asyncId>>>(
				(float *)tile->mCompInfo[0].mCompBuf->mBuf,
				(float *)tile->mCompInfo[1].mCompBuf->mBuf,
				(float *)tile->mCompInfo[2].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,
				result_x_off,
				result_y_off,
				(unsigned char *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
				tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
				bitDepth, 
				bitsPerElement, 
				bitsPerElement,
				elementPerPixel,
				width_pixel,
				threadNum);
#else
			bmi_ict_tile_int32<<<blocknum, threadNum, 0, asyncId>>>(
				(int *)tile->mCompInfo[0].mCompBuf->mBuf,
				(int *)tile->mCompInfo[1].mCompBuf->mBuf,
				(int *)tile->mCompInfo[2].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,
				(int *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
				tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
				bitDepth, 
				width_pixel,
				threadNum);			
#endif
		}
		else
		{
			assert(0); // no done yet
		}

	}
	else		// sync job
	{
		if (wordShift == 2)		// INT_32_BITS
		{
#if GPU_W9X7_FLOAT
			bmi_ict_tile_float<<<blocknum, threadNum, 0>>>(
				(float *)tile->mCompInfo[0].mCompBuf->mBuf,
				(float *)tile->mCompInfo[1].mCompBuf->mBuf,
				(float *)tile->mCompInfo[2].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,
				result_x_off,
				result_y_off,
				(unsigned char *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
				tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
				bitDepth, 
				bitsPerElement, 
				bitsPerElement,
				elementPerPixel,
				width_pixel,
				threadNum);
#else
			bmi_ict_tile_int32<<<blocknum, threadNum, 0>>>(
				(int *)tile->mCompInfo[0].mCompBuf->mBuf,
				(int *)tile->mCompInfo[1].mCompBuf->mBuf,
				(int *)tile->mCompInfo[2].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,
				(int *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
				tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
				bitDepth, 
				width_pixel,
				threadNum);			
#endif
		}
		else
		{
			assert(0); // no done yet
		}

	}
	



	return 0;
}

// int bmi_merge_gpu(TileInfo_c * tile, int decodeLevel, short bitDepth, short wordShift, int red_comp, int green_comp, int blue_comp, EncodeMathod method, StreamId asyncId)
int bmi_merge_gpu(TileInfo_c * tile, int componentNum, int dwtLevel, int decodeLevel, short bitDepth, short wordShift, int inPitchBytes, int elementPerPixel, int bitsPerElement, EncodeMathod method, StreamId asyncId)
{
// 	return 0;

//  	int blocknum = tile->mSize.y;
// 	int width_pixel = tile->mSize.x;
// 
// 	 if (decodeLevel != -1)
// 	{
// 		blocknum = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.y + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.y;
// 		width_pixel = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.x + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.x;
// 	}
// 	int threadNum = width_pixel;
// 	while(threadNum > MAX_THREADS)
// 	{
// 		threadNum = (threadNum + 1) /2;
// 	}




	int threadNum;

	int blocknum = tile->mSize.y; //mCompInfo[0].mCompBuf->mHeight;
	int width_pixel = tile->mSize.x;
	int result_x_off = tile->mOff.x;
	int result_y_off = tile->mOff.y;
	if (decodeLevel < 0 || decodeLevel > dwtLevel)
	{
		decodeLevel = -1;
	}
	if (decodeLevel != -1								// downsize
		&& tile->mCompInfo[0].mNumOfSubband >= decodeLevel * 3)		// not for the thumbnail
	{
		blocknum = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.y + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.y;		
		width_pixel = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.x + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.x;
		while (dwtLevel > decodeLevel)
		{
			result_x_off = (result_x_off + 1) / 2;
			result_y_off = (result_y_off + 1) / 2;
			dwtLevel--;
		}

	}

	threadNum = width_pixel;
	while(threadNum > MAX_THREADS)
	{
		threadNum = (threadNum + 1) /2;
	}

	if (asyncId < 0)	
	{
		asyncId = 0;
	}



	if (wordShift == 2)	
	{
		if (method == Ckernels_W5X3)
		{
			bmi_tile__merge_int32<<<blocknum, threadNum, 0, asyncId>>>(
				(int *)tile->mCompInfo[0].mCompBuf->mBuf,
				(componentNum > 1 ? (int *)tile->mCompInfo[1].mCompBuf->mBuf : NULL),
				(componentNum > 2 ?(int *)tile->mCompInfo[2].mCompBuf->mBuf : NULL),
				(componentNum > 3 ?(int *)tile->mCompInfo[3].mCompBuf->mBuf : NULL),
				tile->mOff.x,
				tile->mOff.y,
				result_x_off,
				result_y_off,
				(unsigned char *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
				tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
				bitDepth, 
				bitsPerElement, 
				bitsPerElement,
				elementPerPixel,
				width_pixel,
				threadNum);
		}
		else
		{

			bmi_tile__merge_float<<<blocknum, threadNum, 0, asyncId>>>(
				(float *)tile->mCompInfo[0].mCompBuf->mBuf,
				(componentNum > 1 ? (float *)tile->mCompInfo[1].mCompBuf->mBuf : NULL),
				(componentNum > 2 ?(float *)tile->mCompInfo[2].mCompBuf->mBuf : NULL),
				(componentNum > 3 ?(float *)tile->mCompInfo[3].mCompBuf->mBuf : NULL),
				tile->mOff.x,
				tile->mOff.y,
				result_x_off,
				result_y_off,
				(unsigned char *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mCompInfo[0].mCompBuf->mPitch, wordShift),
				tile->mResultBuf->mPitch / (elementPerPixel * bitsPerElement / 8), //INT_32_BITS
				bitDepth, 
				bitsPerElement, 
				bitsPerElement,
				elementPerPixel,
				width_pixel,
				threadNum);
		}
	}
	else
	{
		assert(0); // no done yet
	}



	return 0;
}


int bmi_get_one_component(TileInfo_c * tile, int component_id, EncodeMathod method, int hasMCT, int bitDepth, int wordShift, int decodeLevel, StreamId asyncId)
{
	int step = GPU_NUMBER; 
	int blocknum = tile->mSize.y; //mCompInfo[0].mCompBuf->mHeight;
	int width_pixel = tile->mSize.x;
 	if (decodeLevel != -1)
	{
		blocknum = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.y + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.y;
		width_pixel = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.x + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.x;
	}

	if (hasMCT)
	{
		
		if (asyncId > 0)		// async job
		{
			bmi_get_one_component_from_result<<<blocknum, GPU_NUMBER, 0, asyncId>>>(
				component_id,
				tile->mOff.x,
				tile->mOff.y,
				(int *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mResultBuf->mPitch , 2 ), //INT_32_BITS
				width_pixel,
				step);	
		}
		else
		{
			bmi_get_one_component_from_result<<<blocknum, GPU_NUMBER, 0>>>(
				component_id,
				tile->mOff.x,
				tile->mOff.y,
				(int *)tile->mResultBuf->mBuf,		
				BytesToPixels(tile->mResultBuf->mPitch , 2 ), //INT_32_BITS
				width_pixel,
				step);	
		}
	}
	else
	{
		if (method == Ckernels_W5X3)
		{
			if (asyncId > 0)		// async job
			{
				bmi_tile_one_component_int<<<blocknum, GPU_NUMBER, 0, asyncId>>>(
					component_id,
					(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
					tile->mOff.x,
					tile->mOff.y,
					(int *)tile->mResultBuf->mBuf,		
					BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
					BytesToPixels(tile->mResultBuf->mPitch , 2 ), //INT_32_BITS
					bitDepth, 
					width_pixel,
					step);	
			}
			else
			{
				bmi_tile_one_component_int<<<blocknum, GPU_NUMBER, 0>>>(
					component_id,
					(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
					tile->mOff.x,
					tile->mOff.y,
					(int *)tile->mResultBuf->mBuf,		
					BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
					BytesToPixels(tile->mResultBuf->mPitch , 2 ), //INT_32_BITS
					bitDepth, 
					width_pixel,
					step);	
			}
		}
		else	// Ckernels_W9X7
		{
			if (asyncId > 0)		// async job
			{
				bmi_tile_one_component_float<<<blocknum, GPU_NUMBER, 0, asyncId>>>(
					(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
					tile->mOff.x,
					tile->mOff.y,
					(int *)tile->mResultBuf->mBuf,		
					BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
					BytesToPixels(tile->mResultBuf->mPitch , 2 ), //INT_32_BITS
					bitDepth, 
					width_pixel,
					step);		
			}
			else
			{
				bmi_tile_one_component_float<<<blocknum, GPU_NUMBER, 0>>>(
					(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
					tile->mOff.x,
					tile->mOff.y,
					(int *)tile->mResultBuf->mBuf,		
					BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
					BytesToPixels(tile->mResultBuf->mPitch , 2 ), //INT_32_BITS
					bitDepth, 
					width_pixel,
					step);		
			}
		}
	}
	return 0;
}

int bmi_get_one_component_orig(TileInfo_c * tile, int component_id, EncodeMathod method, int bitDepth, int wordShift, int decodeLevel, StreamId asyncId)
{
	int step = GPU_NUMBER; 
	int blocknum = tile->mSize.y; //mCompInfo[0].mCompBuf->mHeight;
	int width_pixel = tile->mSize.x;
 	if (decodeLevel != -1)
	{
		blocknum = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.y + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.y;
		width_pixel = tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 2].mSize.x + tile->mCompInfo[0].mSubbandInfo[decodeLevel * 3 - 1].mSize.x;
	}
	
	if (method == Ckernels_W5X3)
	{
		if (asyncId > 0)		// async job
		{
			bmi_tile_one_component_int_orig<<<blocknum, GPU_NUMBER, 0, asyncId>>>(
				component_id,
				(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,		
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift), //INT_32_BITS
				bitDepth, 
				width_pixel,
				step);	
		}
		else
		{
			bmi_tile_one_component_int_orig<<<blocknum, GPU_NUMBER, 0>>>(
				component_id,
				(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,		
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift), //INT_32_BITS
				bitDepth, 
				width_pixel,
				step);	
		}
	}
	else	// Ckernels_W9X7
	{
		if (asyncId > 0)		// async job
		{
			bmi_tile_one_component_float_orig<<<blocknum, GPU_NUMBER, 0, asyncId>>>(
				(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,	
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift), //INT_32_BITS
				bitDepth, 
				width_pixel,
				step);		
		}
		else
		{
			bmi_tile_one_component_float_orig<<<blocknum, GPU_NUMBER, 0>>>(
				(int *)tile->mCompInfo[component_id].mCompBuf->mBuf,
				tile->mOff.x,
				tile->mOff.y,	
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift),
				BytesToPixels(tile->mCompInfo[component_id].mCompBuf->mPitch, wordShift), //INT_32_BITS
				bitDepth, 
				width_pixel,
				step);		
		}
	}
	
	return 0;
}


#if GPU_T1_TESTING
int gpu_decode_coderblock()
{
	return 0;
}
#endif
